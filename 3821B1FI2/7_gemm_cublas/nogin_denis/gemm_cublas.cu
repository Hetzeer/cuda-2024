// Copyright (c) 2024 Nogin Denis
#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n, 0.0f);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float* d_a;
    float* d_b;
    float* d_c;

    size_t memSize = n * n * sizeof(float);
    
    hipMalloc(&d_a, memSize);
    hipMalloc(&d_b, memSize);
    hipMalloc(&d_c, memSize);

    hipMemcpy(d_a, a.data(), memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), memSize, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 n, n, n,
                 &alpha,
                 d_b, HIP_R_32F, n,
                 d_a, HIP_R_32F, n,
                 &beta,
                 d_c, HIP_R_32F, n,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(c.data(), d_c, memSize, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
