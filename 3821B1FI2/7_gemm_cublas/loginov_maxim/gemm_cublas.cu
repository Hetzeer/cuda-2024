// Copyright (c) 2024 Loginov Maxim
#include <cstdlib>
#include <iostream>
#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gemm_cublas.h"

#define CUDA_CHECK(error) \
    if (error != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUBLAS_CHECK(status) \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error: " << status << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }


std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
  hipDeviceProp_t deviceProp{};
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));

  size_t countElem = n * n;
  if (a.size() != countElem || b.size() != countElem) return {};

  std::vector<float> c(countElem);
  auto bytes = countElem * sizeof(float);
  float alpha = 1.0f;
  float beta = 0.0f;

  float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_a), bytes));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_b), bytes));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_c), bytes));

  CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(d_a), reinterpret_cast<const void*>(a.data()), bytes, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(d_b), reinterpret_cast<const void*>(b.data()), bytes, hipMemcpyHostToDevice));

  hipblasHandle_t handle{};
  CUBLAS_CHECK(hipblasCreate(&handle));
  CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_b, n, d_a, n, &beta, d_c, n));
  CUBLAS_CHECK(hipblasDestroy(handle));

  CUDA_CHECK(hipMemcpy(reinterpret_cast<void*>(c.data()), reinterpret_cast<void*>(d_c), bytes, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_a)));
  CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_b)));
  CUDA_CHECK(hipFree(reinterpret_cast<void*>(d_c)));

  return c;
}