#include "hip/hip_runtime.h"
// Copyright (c) 2024 Loginov Maxim
#include "gelu_cuda.h"
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

__global__ void gelu_kernel(const float* input, float* output, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        float tgH = tanhf(0.7978845608f * (x + 0.044715f * x * x * x));
        output[idx] = 0.5 * x * (1 + tgH);
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t size = input.size();
    size_t bytes = size * sizeof(float);

    std::vector<float> output(size);

    float* d_input = nullptr;
    float* d_output = nullptr;
    hipMalloc((void**)&d_input, bytes);
    hipMalloc((void**)&d_output, bytes);

    hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice);

    int blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int threads = THREADS_PER_BLOCK;

    gelu_kernel<<<blocks, threads>>>(d_input, d_output, size);

    hipMemcpy(output.data(), d_output, bytes, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}