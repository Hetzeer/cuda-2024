#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

#define THREADS_PER_BLOCK 256

__global__ void gelu_kernel(const float* input, float* output, size_t size) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float x = input[i];
        float hyperbolicTan = tanhf(0.7978845608f * (x + 0.044715f * x * x * x));
        output[i] = 0.5 * x * (1 + hyperbolicTan);
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t size = input.size();
    size_t bytes = size * sizeof(float);
    float* d_input = nullptr;
    float* d_output = nullptr;
    std::vector<float> output(size);

    hipMalloc((void**)&d_input, bytes);
    hipMalloc((void**)&d_output, bytes);
    hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice);

    int blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int threads = THREADS_PER_BLOCK;

    gelu_kernel<<<blocks, threads>>>(d_input, d_output, size);
    hipMemcpy(output.data(), d_output, bytes, hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);

    return output;
}