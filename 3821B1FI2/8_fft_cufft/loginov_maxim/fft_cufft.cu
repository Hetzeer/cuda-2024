#include "hip/hip_runtime.h"
// Copyright (c) 2024 Loginov Maxim
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <iostream>

#define CUDA_CHECK(error) \
    if (error != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUFFT_CHECK(status) \
    if (status != HIPFFT_SUCCESS) { \
        std::cerr << "cuFFT Error: " << status << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

__global__ void normalize_kernel(hipfftComplex* data, int total, float inv_n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        data[idx].x *= inv_n;
        data[idx].y *= inv_n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) { 
    if (input.empty()) return {};

    int n = input.size() / (2 * batch);

    std::vector<float> output(input.size(), 0.0f);

    hipfftComplex *d_input = nullptr, *d_output = nullptr;

    size_t bytes = input.size() * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&d_input, bytes));
    CUDA_CHECK(hipMalloc((void**)&d_output, bytes));

    CUDA_CHECK(hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice));

    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlanMany(&plan, 1, &n, NULL, 1, n, NULL, 1, n, HIPFFT_C2C, batch));

    CUFFT_CHECK(hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD));

    CUFFT_CHECK(hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD));

    CUFFT_CHECK(hipfftDestroy(plan));

    float inv_n = 1.0f / static_cast<float>(n);
    int total = n * batch;

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    normalize_kernel<<<blocks, threads>>>(d_input, total, inv_n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(output.data(), d_input, bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    return output;
}