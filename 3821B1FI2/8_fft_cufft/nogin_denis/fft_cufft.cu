#include "hip/hip_runtime.h"
// Copyright (c) 2024 Nogin Denis
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void normKernel(float* input, int size, float norm) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size)
        input[i] *= norm;
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    const int size = input.size();
    std::vector<float> output(size);

    int n = (size / batch) >> 1;
    int memSize = sizeof(hipfftComplex) * n * batch;

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftComplex* data;
    hipMalloc(&data, memSize);
    hipMemcpy(data, input.data(), memSize, hipMemcpyHostToDevice);

    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

    hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    float norm = 1.0f / static_cast<float>(n);
    normKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(data), size, norm);  // hipfftComplex = [float, float]


    hipMemcpy(output.data(), data, memSize, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(data);

    return output;
}
