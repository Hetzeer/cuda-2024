// Copyright (c) 2024 Kokin Ivan

#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

std::vector<float> GemmCUBLAS(const std::vector<float>& a, const std::vector<float>& b, int src) {
  std::vector<float> c(src * src);
  size_t sizeInBytes = src * src * sizeof(*a.data());
  float* device_a;
  float* device_b;
  float* device_c;
  hipMalloc(&device_a, sizeInBytes);
  hipMalloc(&device_b, sizeInBytes);
  hipMalloc(&device_c, sizeInBytes);
  hipMemcpy(device_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, src, src, src, &alpha, device_b, HIP_R_32F, src, device_a, HIP_R_32F, src, &beta, device_c, HIP_R_32F, src, HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);
  hipMemcpy(c.data(), device_c, sizeInBytes, hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  return c;
}
