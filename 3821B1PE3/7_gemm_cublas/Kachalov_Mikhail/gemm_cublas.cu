// Copyright 2024 Kachalov Mikhail
#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float> &a,
                              const std::vector<float> &b,
                              int n)
{
    size_t size = n * n * sizeof(float);
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    float *d_b_t;
    hipMalloc(&d_b_t, size);

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 n, n, n,
                 &alpha,
                 d_b, HIP_R_32F, n,
                 d_a, HIP_R_32F, n,
                 &beta,
                 d_c, HIP_R_32F, n,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_b_t);
    hipFree(d_c);

    hipblasDestroy(handle);

    return c;
}