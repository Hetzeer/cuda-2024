#include "hip/hip_runtime.h"
// Copyright (c) 2025 Ilin Maxim
#include "block_gemm_cuda.h"
#include <hip/hip_runtime.h>


__global__ void BlockGemmKernel(const float* a, const float* b, float* c, int n, int block_size) {
    __shared__ float shared_a[16][16];
    __shared__ float shared_b[16][16];

    int row = blockIdx.y * block_size + threadIdx.y;
    int col = blockIdx.x * block_size + threadIdx.x;
    float sum = 0.0f;

    for (int k = 0; k < n; k += block_size) {
        if (row < n && k + threadIdx.y < n) {
            shared_a[threadIdx.y][threadIdx.x] = a[row * n + k + threadIdx.x];
        } else {
            shared_a[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < n && k + threadIdx.x < n) {
            shared_b[threadIdx.y][threadIdx.x] = b[(k + threadIdx.y) * n + col];
        } else {
            shared_b[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < block_size; ++i) {
            sum += shared_a[threadIdx.y][i] * shared_b[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < n && col < n) {
        c[row * n + col] = sum;
    }
}

std::vector<float> BlockGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    auto countElem = n * n;
    if (a.size() != countElem || b.size() != countElem) return {};

    std::vector<float> c(countElem, 0.0f);

    float* d_a;
    float* d_b;
    float* d_c;
    hipMalloc(&d_a, countElem * sizeof(float));
    hipMalloc(&d_b, countElem * sizeof(float));
    hipMalloc(&d_c, countElem * sizeof(float));

    hipMemcpy(d_a, a.data(), countElem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), countElem * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 16;
    dim3 dimBlock(block_size, block_size);
    dim3 dimGrid((n + block_size - 1) / block_size, (n + block_size - 1) / block_size);

    BlockGemmKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n, block_size);
    hipMemcpy(c.data(), d_c, countElem * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
