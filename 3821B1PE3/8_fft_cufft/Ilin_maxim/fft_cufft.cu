#include "hip/hip_runtime.h"
// Copyright (c) 2025 Ilin Maxim
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void normalizeKernel(float* input, int size, float norm) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        input[i] *= norm;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    const int size = input.size();
    std::vector<float> output(size);

    int n = (size / batch) >> 1;
    int sizeInBytes = sizeof(hipfftComplex) * n * batch;

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftComplex* data;
    hipMalloc(&data, sizeInBytes);
    hipMemcpy(data, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

    hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    float norm = 1.0f / static_cast<float>(n);
    normalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(data), size, norm);


    hipMemcpy(output.data(), data, sizeInBytes, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(data);

    return output;
}
