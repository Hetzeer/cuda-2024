#include "hip/hip_runtime.h"
// Copyright (c) 2024 Fedorets Ilya
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void normalizeKernel(float *input, const int sz, const int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < sz) {
    input[i] /= n;
  }
}

std::vector<float> FffCUFFT(const std::vector<float> &input, int batch) {
  const int sz = input.size();
  std::vector<float> output(sz);
  int n = sz / (batch * 2);

  int sizeInBytes = sizeof(hipfftComplex) * n * batch;
  hipfftHandle plan;
  hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
  hipfftComplex *data;

  hipMalloc(&data, sizeInBytes);
  hipMemcpy(data, input.data(), sizeInBytes, hipMemcpyHostToDevice);
  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
  hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  int threadsPerBlock = deviceProp.maxThreadsPerBlock;
  int blockNum = (input.size() + threadsPerBlock - 1) / threadsPerBlock;
  normalizeKernel<<<blockNum, threadsPerBlock>>>(
      reinterpret_cast<float *>(data), sz, n);

  hipMemcpy(output.data(), data, sizeInBytes, hipMemcpyDeviceToHost);
  hipfftDestroy(plan);
  hipFree(data);
  return output;
}
