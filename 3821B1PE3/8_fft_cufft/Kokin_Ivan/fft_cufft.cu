// Copyright (c) 2024 Kokin Ivan

#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float> &input, int batch)
{
    int src = input.size() / (2 * batch);
    std::vector<float> output(input.size(), 0.0f);

    hipfftHandle plan;
    hipfftComplex *d_input, *d_output;
    hipMalloc((void **)&d_input, input.size() * sizeof(float));
    hipMalloc((void **)&d_output, input.size() * sizeof(float));
    hipMemcpy(d_input, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
    hipfftPlanMany(&plan, 1, &src, NULL, 1, src, NULL, 1, src, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_output, d_output, HIPFFT_BACKWARD);
    hipMemcpy(output.data(), d_output, output.size() * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < output.size(); i++)
    {
        output[i] /= src;
    }

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);

    return output;
}