#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>


__global__ void NaiveGemmKernel(const float* a,
                       const float* b,
                       float* c,
                       int n) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    int elemCount = n * n;
    if (a.size() != elemCount || b.size() != elemCount) return {};

    std::vector<float> c(elemCount, 0.0f);

    float* d_a;
    float* d_b;
    float* d_c;

    hipMalloc(&d_a, n * n * sizeof(float));
    hipMalloc(&d_b, n * n * sizeof(float));
    hipMalloc(&d_c, n * n * sizeof(float));

    hipMemcpy(d_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    const size_t axSize = 32u;
    dim3 threadsPerBlock(
        axSize,
        axSize
    );
    dim3 numBlocks(
        (n + axSize - 1) / axSize,
        (n + axSize - 1) / axSize
    );

    NaiveGemmKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(c.data(), d_c, elemCount * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
