#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gelu_cuda.h"


__global__ void GeluKernel(const float* input, float* output, size_t size) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= size) return;

  constexpr float geluCoeff1 = 1.595769122f;
  constexpr float geluCoeff2 = 0.071354816f;

  float value = input[i];
  output[i] = value * (1 - 1 / (1.0f + __expf(value * (geluCoeff1 + value * value * geluCoeff2))));
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  if (input.empty()) return {};

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  auto size = input.size();
  std::vector<float> output(size);

  auto sizeInBytes = size * sizeof(float);
  auto threadsPerBlock = deviceProp.maxThreadsPerBlock;
  auto numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

  float *inputDev = nullptr;
  hipMalloc(&inputDev, sizeInBytes);
  
  float *outputDev = nullptr;
  hipMalloc(&outputDev, sizeInBytes);

  hipMemcpy(inputDev, input.data(), sizeInBytes, hipMemcpyHostToDevice);

  GeluKernel<<<numBlocks, threadsPerBlock>>>(inputDev, outputDev, size);

  hipDeviceSynchronize();
  hipMemcpy(output.data(), outputDev, sizeInBytes, hipMemcpyDeviceToHost);

  hipFree(outputDev);
  hipFree(inputDev);
  return output;
}
