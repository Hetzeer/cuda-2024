#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include ""
#include "fft_cufft.h"

#define CHECK_CUDA_ERROR(callable)                                        \
  {                                                                       \
    auto codeError = callable;                                            \
    if (codeError != hipSuccess) {                                       \
      std::cerr << "\033[1;31merror\033[0m: ";                            \
      std::cerr << hipGetErrorString(codeError) << '\n';                 \
      std::cerr << "code error: " << static_cast<int>(codeError) << '\n'; \
      std::cerr << "loc: " << __FILE__ << '(' << __LINE__ << ")\n";       \
      std::exit(static_cast<int>(codeError));                             \
    }                                                                     \
  }

#define CHECK_CUFFT_RESULT(callable)                                \
  {                                                                 \
    auto result = callable;                                         \
    if (result != HIPFFT_SUCCESS) {                                  \
      std::cerr << "\033[1;31mcufft result failed:\033[0m: ";       \
      std::cerr << static_cast<int>(result) << '\n';                \
      std::cerr << "loc: " << __FILE__ << '(' << __LINE__ << ")\n"; \
      std::exit(static_cast<int>(result));                          \
    }                                                               \
  }

__global__ void normalize_kernel(float* x, size_t size, float coef) {
  size_t i = static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) +
             static_cast<size_t>(threadIdx.x);
  if (i < size) {
    x[i] *= coef;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  hipDeviceProp_t deviceProp{};
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&deviceProp, 0));

  if (input.empty()) return {};

  auto size = input.size();
  auto elemPerBatch = size / batch >> 1;
  auto countBytes = sizeof(hipfftComplex) * elemPerBatch * batch;
  auto threadsPerBlock = deviceProp.maxThreadsPerBlock;
  auto numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
  std::vector<float> output(size);

  hipfftComplex* signal = nullptr;
  CHECK_CUDA_ERROR(hipMalloc(&signal, countBytes));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void*>(signal),
                              reinterpret_cast<const void*>(input.data()),
                              countBytes, hipMemcpyHostToDevice));

  hipfftHandle handle{};
  CHECK_CUFFT_RESULT(hipfftPlan1d(&handle, elemPerBatch, HIPFFT_C2C, batch));
  CHECK_CUFFT_RESULT(hipfftExecC2C(handle, signal, signal, HIPFFT_FORWARD));
  CHECK_CUFFT_RESULT(hipfftExecC2C(handle, signal, signal, HIPFFT_BACKWARD));
  CHECK_CUFFT_RESULT(hipfftDestroy(handle));

  normalize_kernel<<<numBlocks, threadsPerBlock>>>(
      reinterpret_cast<float*>(signal), size, 1.0f / elemPerBatch);
  CHECK_CUDA_ERROR(hipDeviceSynchronize());
  CHECK_CUDA_ERROR(hipGetLastError());

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void*>(output.data()),
                              reinterpret_cast<void*>(signal), countBytes,
                              hipMemcpyDeviceToHost));
  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void*>(signal)));

  return output;
}
