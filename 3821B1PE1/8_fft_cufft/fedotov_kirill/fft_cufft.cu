#include "hip/hip_runtime.h"
// Copyright (c) 2024 Fedotov Kirill
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>

__global__ void normalizeKernel(float* in, int size, float normal) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        in[i] *= normal;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    const int size = input.size();
    std::vector<float> output(size);
    int n = (size / batch) >> 1;\

    int sizeInBytes = sizeof(hipfftComplex) * n * batch;
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
    hipfftComplex* data;

    hipMalloc(&data, sizeInBytes);
    hipMemcpy(data, input.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    float norm = 1.0f / static_cast<float>(n);
    normalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(data), size, norm);
		
    hipMemcpy(output.data(), data, sizeInBytes, hipMemcpyDeviceToHost);
    hipfftDestroy(plan);
    hipFree(data);
    return output;
}