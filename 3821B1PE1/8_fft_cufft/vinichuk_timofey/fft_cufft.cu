#include "hip/hip_runtime.h"
// Copyright (c) 2024 Vinichuk Timofey
#include "fft_cufft.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

__global__ void NormalizeKernel(float* input, int size, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        input[index] /= N;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    std::vector<float> result(input.size());

    int N = input.size() / (batch * 2);

    hipfftHandle handle;
    hipfftComplex* data;

    hipfftPlan1d(&handle, N, HIPFFT_C2C, batch);

    hipMalloc(&data, sizeof(hipfftComplex) * N * batch);
    hipMemcpy(
        data,
        input.data(),
        sizeof(hipfftComplex) * N * batch,
        hipMemcpyHostToDevice
    );

    hipfftExecC2C(handle, data, data, HIPFFT_FORWARD);
    hipfftExecC2C(handle, data, data, HIPFFT_BACKWARD);

    hipDeviceProp_t devPropts;
    hipGetDeviceProperties(&devPropts, 0);
    size_t threadsPerBlock = devPropts.maxThreadsPerBlock;
    size_t blocksCount = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

    NormalizeKernel << <blocksCount, threadsPerBlock >> > (
        reinterpret_cast<float*>(data),
        result.size(),
        N
        );

    hipMemcpy(
        result.data(),
        data,
        sizeof(hipfftComplex) * N * batch,
        hipMemcpyDeviceToHost
    );

    hipfftDestroy(handle);
    hipFree(data);

    return result;
}