// Copyright (c) 2024 Fedotov Kirill
#include <cstdlib>
#include <iostream>

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gemm_cublas.h"

#define CHECK_CUDA_ERROR(callable)                                        \
  {                                                                       \
    auto codeError = callable;                                            \
    if (codeError != hipSuccess) {                                       \
      std::cerr << "\033[1;31merror\033[0m: ";                            \
      std::cerr << hipGetErrorString(codeError) << '\n';                 \
      std::cerr << "code error: " << static_cast<int>(codeError) << '\n'; \
      std::cerr << "loc: " << __FILE__ << '(' << __LINE__ << ")\n";       \
      std::exit(codeError);                                               \
    }                                                                     \
  }

#define CHECK_CUBLAS_STATUS(callable)                               \
  {                                                                 \
    auto status = callable;                                         \
    if (status != HIPBLAS_STATUS_SUCCESS) {                          \
      std::cerr << "\033[1;31mcublas status failed:\033[0m: ";      \
      std::cerr << static_cast<int>(status) << '\n';                \
      std::cerr << "loc: " << __FILE__ << '(' << __LINE__ << ")\n"; \
      std::exit(status);                                            \
    }                                                               \
  }

std::vector<float> GemmCUBLAS(const std::vector<float>& matrix_a, const std::vector<float>& matrix_b, int matrix_size) {
  hipDeviceProp_t device_properties{};
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&device_properties, 0));

  size_t total_elements = matrix_size * matrix_size;
  if (matrix_a.size() != total_elements || matrix_b.size() != total_elements) return {};

  std::vector<float> matrix_c_host(total_elements);
  auto total_bytes = total_elements * sizeof(float);
  float alpha = 1.0f;
  float beta = 0.0f;

  float* matrix_a_device = nullptr;
  float* matrix_b_device = nullptr;
  float* matrix_c_device = nullptr;

  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&matrix_a_device), total_bytes));
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&matrix_b_device), total_bytes));
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void**>(&matrix_c_device), total_bytes));

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void*>(matrix_a_device), reinterpret_cast<const void*>(matrix_a.data()), total_bytes, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void*>(matrix_b_device), reinterpret_cast<const void*>(matrix_b.data()), total_bytes, hipMemcpyHostToDevice));

  hipblasHandle_t cublas_handle{};
  CHECK_CUBLAS_STATUS(hipblasCreate(&cublas_handle));
  CHECK_CUBLAS_STATUS(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size, matrix_size, matrix_size, &alpha, matrix_b_device, matrix_size, matrix_a_device, matrix_size, &beta, matrix_c_device, matrix_size));
  CHECK_CUBLAS_STATUS(hipblasDestroy(cublas_handle));

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void*>(matrix_c_host.data()), reinterpret_cast<void*>(matrix_c_device), total_bytes, hipMemcpyDeviceToHost));

  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void*>(matrix_a_device)));
  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void*>(matrix_b_device)));
  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void*>(matrix_c_device)));

  return matrix_c_host;
}
