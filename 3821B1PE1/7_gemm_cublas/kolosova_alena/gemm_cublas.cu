#include <iostream>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "gemm_cublas.h"


std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b, 
                              int size) 
{
    const float alpha = 1.0f;
    const float beta = 0.0f;
    const auto countElem = size * size;
    std::vector<float> output(countElem);
    const auto sizeInBytes = countElem * sizeof(float);

    float *aDev = nullptr;
    hipMalloc(&aDev, sizeInBytes);
    float *bDev = nullptr;
    hipMalloc(&bDev, sizeInBytes);
    float *cDev = nullptr;
    hipMalloc(&cDev, sizeInBytes);

    hipMemcpy(aDev, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(bDev, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha,
                 bDev, HIP_R_32F, size,
                 aDev, HIP_R_32F, size,
                 &beta,
                 cDev, HIP_R_32F, size,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(output.data(), cDev, sizeInBytes, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(cDev);
    hipFree(bDev);
    hipFree(aDev);

    return output;
}
