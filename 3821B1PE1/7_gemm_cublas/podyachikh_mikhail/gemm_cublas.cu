// Copyright (c) 2024 Podyachikh Mikhail
#include "gemm_cublas.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>

std::vector<float> GemmCUBLAS(const std::vector<float> &a,
                              const std::vector<float> &b,
                              int n) {
  std::vector<float> c(n * n);

  float *buf_a, *buf_b, *buf_c;
  hipMalloc(&buf_a, n * n * sizeof(float));
  hipMalloc(&buf_b, n * n * sizeof(float));
  hipMalloc(&buf_c, n * n * sizeof(float));

  hipMemcpy(buf_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(buf_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, buf_b,
              n, buf_a, n, &beta, buf_c, n);
  hipblasDestroy(handle);

  hipMemcpy(c.data(), buf_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(buf_a);
  hipFree(buf_b);
  hipFree(buf_c);

  return c;
}
