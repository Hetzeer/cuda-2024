#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b, int n) {
    std::vector<float> c(n * n);

    size_t sizeInBytes = n * n * sizeof(float);

    float* device_a;
    float* device_b;
    float* device_c;

    if (hipMalloc(&device_a, sizeInBytes) != hipSuccess) {
        std::cerr << "Error allocating device memory for A" << std::endl;
        return c;
    }
    if (hipMalloc(&device_b, sizeInBytes) != hipSuccess) {
        std::cerr << "Error allocating device memory for B" << std::endl;
        hipFree(device_a);
        return c;
    }
    if (hipMalloc(&device_c, sizeInBytes) != hipSuccess) {
        std::cerr << "Error allocating device memory for C" << std::endl;
        hipFree(device_a);
        hipFree(device_b);
        return c;
    }

    if (hipMemcpy(device_a, a.data(), sizeInBytes, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error copying A to device" << std::endl;
        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);
        return c;
    }
    if (hipMemcpy(device_b, b.data(), sizeInBytes, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error copying B to device" << std::endl;
        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);
        return c;
    }

    hipblasHandle_t handle;
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "Error creating cuBLAS handle" << std::endl;
        hipFree(device_a);
        hipFree(device_b);
        hipFree(device_c);
        return c;
    }

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    if (hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
                     &alpha, device_b, HIP_R_32F, n,
                     device_a, HIP_R_32F, n,
                     &beta, device_c, HIP_R_32F, n,
                     HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT) != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "Error in cuBLAS matrix multiplication" << std::endl;
    }

    if (hipMemcpy(c.data(), device_c, sizeInBytes, hipMemcpyDeviceToHost) != hipSuccess) {
        std::cerr << "Error copying C from device to host" << std::endl;
    }

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    hipblasDestroy(handle);

    return c;
}
