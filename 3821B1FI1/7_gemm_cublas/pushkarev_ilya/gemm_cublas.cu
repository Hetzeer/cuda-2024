#include <hip/hip_runtime.h>

#include "gemm_cublas.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) 
{
    const float q = 1.0f;
    const float p = 0.0f;

    std::vector<float> c(n * n);

    float* deviceA;
    float* deviceB;
    float* deviceC;

    hipMalloc(&deviceA, sizeof(float) * n * n);
    hipMalloc(&deviceB, sizeof(float) * n * n);
    hipMalloc(&deviceC, sizeof(float) * n * n);

    hipMemcpy(deviceA, a.data(), sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b.data(), sizeof(float) * n * n, hipMemcpyHostToDevice);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &q, deviceB, n, deviceA, n, &p, deviceC, n);

    hipblasDestroy(cublasHandle);

    hipMemcpy(c.data(), deviceC, sizeof(float) * n * n, hipMemcpyDeviceToHost);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return c;
}