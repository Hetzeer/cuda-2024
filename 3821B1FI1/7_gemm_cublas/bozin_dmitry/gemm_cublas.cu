// Copyright (c) 2024 Bozin Dmitry
#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a, const std::vector<float>& b, int n) {
  const unsigned _n = n;
  const unsigned sz = _n * _n;
  const unsigned sz_bytes = sz * sizeof(float);
  if (a.size() != sz || b.size() != sz) {
    return std::vector<float>();
  }
  std::vector<float> res(sz);
  float* a_dev;
  float* b_dev;
  float* res_dev;
  hipMalloc((void**)&a_dev, sz_bytes);
  hipMalloc((void**)&b_dev, sz_bytes);
  hipMalloc((void**)&res_dev, sz_bytes);
  hipMemcpy(a_dev, a.data(), sz_bytes, hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b.data(), sz_bytes, hipMemcpyHostToDevice);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  constexpr float alpha = 1.0f;
  constexpr float beta = 0.0f;
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha,
    b_dev, HIP_R_32F, n,
    a_dev, HIP_R_32F, n,
    &beta, res_dev, HIP_R_32F, n, HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);
  hipMemcpy(res.data(), res_dev, sz_bytes, hipMemcpyDeviceToHost);
  return res;
}