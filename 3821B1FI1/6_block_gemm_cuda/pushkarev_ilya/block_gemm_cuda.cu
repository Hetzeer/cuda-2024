#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>

#include "block_gemm_cuda.h"
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 32

__global__ void BlockGemmKernel(const float* a, const float* b,
                                float* const c, const int size) 
{

    __shared__ float aCached[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float bCached[BLOCK_SIZE][BLOCK_SIZE];

    const int threadX = threadIdx.x;
    const int threadY = threadIdx.y;

    const int rowIndex = blockIdx.y * BLOCK_SIZE + threadY;
    const int colIndex = blockIdx.x * BLOCK_SIZE + threadX;

    float resultValue = 0.0f;

    for (int tile = 0; tile < size / BLOCK_SIZE; ++tile) 
    {
        aCached[threadY][threadX] = a[rowIndex * size + tile * BLOCK_SIZE + threadX];
        bCached[threadY][threadX] = b[(tile * BLOCK_SIZE + threadY) * size + colIndex];

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            resultValue += aCached[threadY][k] * bCached[k][threadX];
        }
        __syncthreads();
    }

    if (rowIndex < size && colIndex < size) 
    {
        c[rowIndex * size + colIndex] = resultValue;
    }
}

std::vector<float> BlockGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b, 
                                 int n) 
{
    std::vector<float> resultMatrix(n * n);

    size_t sizeInBytes = n * n * sizeof(*a.data());

    float* deviceMatrixA;
    hipMalloc(&deviceMatrixA, sizeInBytes);
    float* deviceMatrixB;
    hipMalloc(&deviceMatrixB, sizeInBytes);
    float* deviceResultMatrix;
    hipMalloc(&deviceResultMatrix, sizeInBytes);

    hipMemcpy(deviceMatrixA, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    const int blockSizeAxis = BLOCK_SIZE;
    dim3 threadsPerBlock(blockSizeAxis, blockSizeAxis);
    dim3 numBlocks((n + blockSizeAxis - 1) / blockSizeAxis, (n + blockSizeAxis - 1) / blockSizeAxis);

    BlockGemmKernel<<<numBlocks, threadsPerBlock>>>(deviceMatrixA, deviceMatrixB, deviceResultMatrix, n);

    hipMemcpy(resultMatrix.data(), deviceResultMatrix, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceResultMatrix);
    return resultMatrix;
}