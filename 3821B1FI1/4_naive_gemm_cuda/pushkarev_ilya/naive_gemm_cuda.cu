#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include "hip/hip_runtime.h"


__global__ void naiveGemmKernel(const float* a, const float* b, float* c, int n) 
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < n) 
    {
        float sum = 0.0f;
        for (int r = 0; r < n; ++r) 
        {
            sum += a[i * n + r] * b[r * n + j];
        }
        c[i * n + j] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) 
{
    std::vector<float> c(n * n, 0.0f);

    float* d_a;
    float* d_b;
    float* d_c;

    hipMalloc(&d_a, n * n * sizeof(float));
    hipMalloc(&d_b, n * n * sizeof(float));
    hipMalloc(&d_c, n * n * sizeof(float));
    hipMemcpy(d_a, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    const size_t blockSize = 32u;
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 gridDimensions((n + blockSize - 1) / blockSize, (n + blockSize - 1) / blockSize);

    naiveGemmKernel<<<gridDimensions, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(c.data(), d_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}