/*When entering the following vector:
int batch = 1;
    std::vector<float> input = {1.0, 0.0, 2.0, 0.0, 3.0, 0.0, 4.0, 0.0}; 
The output values ​​were:
1 0 2 0 3 0 4 0 
*/

#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);

    hipfftComplex* d_input;
    hipfftComplex* d_output;
    std::vector<float> output(2 * n * batch);

    hipMalloc((void**)&d_input, sizeof(hipfftComplex) * n * batch);
    hipMalloc((void**)&d_output, sizeof(hipfftComplex) * n * batch);

    hipMemcpy(d_input, input.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);

    hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD);

    float normalizationFactor = 1.0f / n;
    hipMemcpy(output.data(), d_input, sizeof(float) * output.size(), hipMemcpyDeviceToHost);
    for (int i = 0; i < output.size(); ++i) {
        output[i] *= normalizationFactor;
    }

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);

    return output;
}
