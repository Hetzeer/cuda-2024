#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <iostream>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);

    hipfftComplex* d_input;
    hipfftComplex* d_output;
    std::vector<float> output(2 * n * batch);

    hipMalloc((void**)&d_input, sizeof(hipfftComplex) * n * batch);
    hipMalloc((void**)&d_output, sizeof(hipfftComplex) * n * batch);

    hipMemcpy(d_input, input.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD);

    hipfftExecC2C(plan, d_output, d_input, HIPFFT_BACKWARD);

    float normalizationFactor = 1.0f / n;
    hipMemcpy(output.data(), d_input, sizeof(float) * output.size(), hipMemcpyDeviceToHost);
    for (int i = 0; i < output.size(); ++i) {
        output[i] *= normalizationFactor;
    }

    hipfftDestroy(plan);
    hipFree(d_input);
    hipFree(d_output);

    return output;
}
