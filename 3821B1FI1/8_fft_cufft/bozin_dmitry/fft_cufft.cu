#include "hip/hip_runtime.h"
// Copyright (c) 2024 Bozin Dmitry
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <cassert>

__constant__ float input_norm;

__global__ void norm_kernel(float* data, const unsigned n) {
  unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    data[i] *= input_norm;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  const unsigned n = input.size();
  assert(n % (batch + batch) == 0);
  std::vector<float> res(n);
  const unsigned n_batch = n / batch >> 1;
  const unsigned n_bytes = sizeof(hipfftComplex) * n_batch * batch;
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, device);
  const unsigned block_size = device_prop.maxThreadsPerBlock;
  const unsigned num_blocks = (n + block_size - 1) / block_size;
  const float norm = 1.0f / static_cast<float>(n_batch);
  hipMemcpyToSymbol(HIP_SYMBOL(input_norm), &norm, sizeof(norm));
  hipfftComplex* data;
  hipMalloc(&data, n_bytes);
  hipMemcpy(data, input.data(), n_bytes, hipMemcpyHostToDevice);
  hipfftHandle handle;
  hipfftPlan1d(&handle, n_batch, HIPFFT_C2C, batch);
  hipfftExecC2C(handle, data, data, HIPFFT_FORWARD);
  hipfftExecC2C(handle, data, data, HIPFFT_BACKWARD);
  norm_kernel<<<num_blocks, block_size>>>(reinterpret_cast<float*>(data), n);
  hipMemcpy(res.data(), data, n_bytes, hipMemcpyDeviceToHost);
  return res;
}