#include <hipfft/hipfft.h>
#include <iostream>

#include "fft_cufft.h"

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) 
{
    int n = input.size() / (2 * batch);

    hipfftComplex* device_input;
    hipfftComplex* device_output;

    std::vector<float> output(2 * n * batch);

    hipMalloc((void**)&device_input, sizeof(hipfftComplex) * n * batch);
    hipMalloc((void**)&device_output, sizeof(hipfftComplex) * n * batch);

    hipMemcpy(device_input, input.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);

    hipfftHandle fft_plan;

    hipfftPlan1d(&fft_plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(fft_plan, device_input, device_output, HIPFFT_FORWARD);
    hipfftExecC2C(fft_plan, device_output, device_input, HIPFFT_BACKWARD);

    float norm_factor = 1.0f / n;

    hipMemcpy(output.data(), device_input, sizeof(float) * output.size(), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < output.size(); ++i) {
        output[i] *= norm_factor;
    }

    hipfftDestroy(fft_plan);
    hipFree(device_input);
    hipFree(device_output);

    return output;
}