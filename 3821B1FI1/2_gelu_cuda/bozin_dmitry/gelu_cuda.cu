#include "hip/hip_runtime.h"
// Copyright (c) 2024 Bozin Dmitry
#include "gelu_cuda.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<random>
#include<chrono>


__global__ void GelKern(const float* input, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr float sqrt2_pi = 0.797885f;
    constexpr float coeff = 0.044715f;
    if (idx < size) {
        float x = input[idx]; 
        float tanh_arg = sqrt2_pi * (x + coeff * x * x * x);
        output[idx] = 0.5f * x * (1.0f + tanh(tanh_arg));
    }
}



std::vector<float> GeluCUDA(const std::vector<float>& input) {
    const size_t vectorSize = input.size();
    const int blockSize = 256;
    int numBlocks = (vectorSize + blockSize - 1) / blockSize;
    std::vector<float>result(vectorSize);
    float* deviceInput = nullptr;
    float* deviceOutput = nullptr;
    hipMalloc(&deviceInput,vectorSize * sizeof(float));
    hipMalloc(&deviceOutput,vectorSize * sizeof(float));
    hipMemcpy(deviceInput, input.data(), vectorSize * sizeof(float), hipMemcpyHostToDevice);
    GelKern<<<numBlocks, blockSize>>>(deviceInput, deviceOutput, vectorSize);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
    hipMemcpy(result.data(), deviceOutput, vectorSize * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(deviceInput);
    hipFree(deviceOutput);

    return result;
}
