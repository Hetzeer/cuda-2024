#include "hip/hip_runtime.h"
#include <cmath>

#include "gelu_cuda.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void geluKernel(const float* input, float* output, size_t size_input) 
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    constexpr float temp = 0.7978845608f;

    if (i < size_input) 
    {
        float x = input[i];
        output[i] = 0.5f * x * (1.f + tanhf(temp * (x + 0.044715f * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) 
{
    auto size_input = input.size();

    if (size_input == 0) 
    {
        return {};
    }

    float* d_input;
    float* d_output;
    std::vector<float> result(size_input);
    hipMalloc(&d_input, size_input * sizeof(float));
    hipMalloc(&d_output, size_input * sizeof(float));

    hipMemcpy(d_input, input.data(), size_input * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size_input + threadsPerBlock - 1) / threadsPerBlock;
    geluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size_input);

    hipMemcpy(result.data(), d_output, size_input * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);

    return result;
}