#include "hip/hip_runtime.h"
// Copyright (c) 2024 Ulyanov Daniil

#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "naive_gemm_cuda.h"

__global__ void MatrixMulKernel(const float *a, const float *b, float *c,
                                int n) {
  int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (rowIdx < n && colIdx < n) {
    float partialSum = 0.0f;
    for (int idx = 0; idx < n; ++idx) {
      partialSum += a[rowIdx * n + idx] * b[idx * n + colIdx];
    }
    c[rowIdx * n + colIdx] = partialSum;
  }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float> &a,
                                 const std::vector<float> &b, int n) {
  std::vector<float> resultMatrix(n * n);

  float *deviceA = nullptr;
  float *deviceB = nullptr;
  float *deviceC = nullptr;

  hipMalloc(&deviceA, static_cast<size_t>(n) * n * sizeof(float));
  hipMalloc(&deviceB, static_cast<size_t>(n) * n * sizeof(float));
  hipMalloc(&deviceC, static_cast<size_t>(n) * n * sizeof(float));

  hipMemcpy(deviceA, a.data(), static_cast<size_t>(n) * n * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceB, b.data(), static_cast<size_t>(n) * n * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (n + threadsPerBlock.y - 1) / threadsPerBlock.y);

  MatrixMulKernel<<<numBlocks, threadsPerBlock>>>(deviceA, deviceB, deviceC, n);

  hipMemcpy(resultMatrix.data(), deviceC,
             static_cast<size_t>(n) * n * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  return resultMatrix;
}
