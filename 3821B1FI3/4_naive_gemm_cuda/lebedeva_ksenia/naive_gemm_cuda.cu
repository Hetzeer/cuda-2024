#include "hip/hip_runtime.h"
// Copyright (c) 2024 Lebedeva Ksenia
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "naive_gemm_cuda.h"

#define CHECK_CUDA_ERROR(callable)                                        \
  {                                                                       \
    auto codeError = callable;                                            \
    if (codeError != hipSuccess) {                                       \
      std::cerr << "\033[1;31merror\033[0m: ";                            \
      std::cerr << hipGetErrorString(codeError) << '\n';                 \
      std::cerr << "code error: " << static_cast<int>(codeError) << '\n'; \
      std::cerr << "loc: " << __FILE__ << '(' << __LINE__ << ")\n";       \
      std::exit(codeError);                                               \
    }                                                                     \
  }

constexpr auto BLOCK_SIZE = 32;

__global__ void naive_gemm_kernel(float *c, const float *a, const float *b,
                                  const size_t size) {
  size_t iIdx = blockIdx.y * blockDim.y + threadIdx.y;
  size_t jIdx = blockIdx.x * blockDim.x + threadIdx.x;

  if (iIdx < size && jIdx < size) {
    float resCell{};
    for (size_t i = 0; i < size; ++i)
      resCell += a[iIdx * size + i] * b[size * i + jIdx];
    c[iIdx * size + jIdx] = resCell;
  }
}

static constexpr int cdiv(int a, int b) noexcept { return (a + b - 1) / b; }

std::vector<float> NaiveGemmCUDA(const std::vector<float> &a,
                                 const std::vector<float> &b, int size) {
  hipDeviceProp_t deviceProp{};
  CHECK_CUDA_ERROR(hipGetDeviceProperties(&deviceProp, 0));

  size_t countElem = size * size;
  if (a.size() != countElem || b.size() != countElem) return {};

  std::vector<float> cHost(countElem);
  auto countBytes = countElem * sizeof(float);
  constexpr auto sizeAxis = BLOCK_SIZE;
  dim3 threadsPerBlock(sizeAxis, sizeAxis);
  dim3 numBlocks(cdiv(size, sizeAxis), cdiv(size, sizeAxis));

  float *aDev = nullptr;
  float *bDev = nullptr;
  float *cDev = nullptr;

  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&aDev), countBytes));
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&bDev), countBytes));
  CHECK_CUDA_ERROR(hipMalloc(reinterpret_cast<void **>(&cDev), countBytes));

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(aDev),
                              reinterpret_cast<const void *>(a.data()),
                              countBytes, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(bDev),
                              reinterpret_cast<const void *>(b.data()),
                              countBytes, hipMemcpyHostToDevice));

  naive_gemm_kernel<<<numBlocks, threadsPerBlock>>>(cDev, aDev, bDev, size);
  CHECK_CUDA_ERROR(hipDeviceSynchronize());
  CHECK_CUDA_ERROR(hipGetLastError());

  CHECK_CUDA_ERROR(hipMemcpy(reinterpret_cast<void *>(cHost.data()),
                              reinterpret_cast<void *>(cDev), countBytes,
                              hipMemcpyDeviceToHost));

  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void *>(aDev)));
  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void *>(bDev)));
  CHECK_CUDA_ERROR(hipFree(reinterpret_cast<void *>(cDev)));

  return cHost;
}
