#include "hip/hip_runtime.h"
// Copyright (c) 2024 Ulyanov Daniil

#include <hip/hip_runtime.h>
#include <>

#include <cmath>
#include <iostream>
#include <vector>

#include "gelu_cuda.h"

__global__ void GeluKernel(const float *input, float *output, int size) {
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId < size) {
    float value = input[threadId];
    float gelu_cdf =
        0.5f * (1.0f + tanhf(0.7978845608028654f *
                             (value + 0.044715f * value * value * value)));
    output[threadId] = value * gelu_cdf;
  }
}

std::vector<float> GeluCUDA(const std::vector<float> &input) {
  if (input.empty())
    return {};

  hipDeviceProp_t gpuProperties;
  hipGetDeviceProperties(&gpuProperties, 0);

  size_t arraySize = input.size();
  size_t byteSize = arraySize * sizeof(float);

  std::vector<float> hostOutput(arraySize);

  int threadsInBlock = gpuProperties.maxThreadsPerBlock;
  int blocksCount = (arraySize + threadsInBlock - 1) / threadsInBlock;

  float *deviceInput = nullptr;
  float *deviceOutput = nullptr;

  hipMalloc(&deviceInput, byteSize);
  hipMalloc(&deviceOutput, byteSize);

  hipMemcpy(deviceInput, input.data(), byteSize, hipMemcpyHostToDevice);

  GeluKernel<<<blocksCount, threadsInBlock>>>(deviceInput, deviceOutput,
                                              arraySize);

  hipMemcpy(hostOutput.data(), deviceOutput, byteSize, hipMemcpyDeviceToHost);

  hipFree(deviceInput);
  hipFree(deviceOutput);

  return hostOutput;
}
