// Copyright (c) 2024 Lebedeva Ksenia
#include "gemm_cublas.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <cstdlib>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b, int size) {
    std::vector<float> c(size * size);

    size_t sizeInBytes = size * size * sizeof(*a.data());

    float* d_a;
    hipMalloc(&d_a, sizeInBytes);
    float* d_b;
    hipMalloc(&d_b, sizeInBytes);
    float* d_c;
    hipMalloc(&d_c, sizeInBytes);

    hipMemcpy(d_a, a.data(), sizeInBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha,
                 d_b, HIP_R_32F, size,
                 d_a, HIP_R_32F, size,
                 &beta,
                 d_c, HIP_R_32F, size,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    hipMemcpy(c.data(), d_c, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return c;
}
