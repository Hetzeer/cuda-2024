// Copyright (c) 2024 Ulyanov Daniil

#include <cstdlib>
#include <iostream>

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gemm_cublas.h"

#define VERIFY_CUDA_CALL(call)                                                 \
  {                                                                            \
    auto errorCode = call;                                                     \
    if (errorCode != hipSuccess) {                                            \
      std::cerr << "\033[1;31mCUDA Error:\033[0m ";                            \
      std::cerr << hipGetErrorString(errorCode) << '\n';                      \
      std::cerr << "Error code: " << static_cast<int>(errorCode) << '\n';      \
      std::cerr << "Location: " << __FILE__ << " (" << __LINE__ << ")\n";      \
      std::exit(errorCode);                                                    \
    }                                                                          \
  }

#define VERIFY_CUBLAS_CALL(call)                                               \
  {                                                                            \
    auto hipblasStatus_t = call;                                                  \
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {                               \
      std::cerr << "\033[1;31mcuBLAS Error:\033[0m ";                          \
      std::cerr << static_cast<int>(hipblasStatus_t) << '\n';                     \
      std::cerr << "Location: " << __FILE__ << " (" << __LINE__ << ")\n";      \
      std::exit(hipblasStatus_t);                                                 \
    }                                                                          \
  }

std::vector<float> GemmCUBLAS(const std::vector<float> &a,
                              const std::vector<float> &b, int size) {
  hipDeviceProp_t deviceProperties{};
  VERIFY_CUDA_CALL(hipGetDeviceProperties(&deviceProperties, 0));

  size_t totalElements = size * size;
  if (a.size() != totalElements || b.size() != totalElements)
    return {};

  std::vector<float> resultHost(totalElements);
  auto totalBytes = totalElements * sizeof(float);
  float alphaVal = 1.0f;
  float betaVal = 0.0f;

  float *devA = nullptr;
  float *devB = nullptr;
  float *devC = nullptr;

  VERIFY_CUDA_CALL(hipMalloc(reinterpret_cast<void **>(&devA), totalBytes));
  VERIFY_CUDA_CALL(hipMalloc(reinterpret_cast<void **>(&devB), totalBytes));
  VERIFY_CUDA_CALL(hipMalloc(reinterpret_cast<void **>(&devC), totalBytes));

  VERIFY_CUDA_CALL(hipMemcpy(reinterpret_cast<void *>(devA),
                              reinterpret_cast<const void *>(a.data()),
                              totalBytes, hipMemcpyHostToDevice));
  VERIFY_CUDA_CALL(hipMemcpy(reinterpret_cast<void *>(devB),
                              reinterpret_cast<const void *>(b.data()),
                              totalBytes, hipMemcpyHostToDevice));

  hipblasHandle_t cublasHandle{};
  VERIFY_CUBLAS_CALL(hipblasCreate(&cublasHandle));
  VERIFY_CUBLAS_CALL(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, size,
                                 size, size, &alphaVal, devB, size, devA, size,
                                 &betaVal, devC, size));
  VERIFY_CUBLAS_CALL(hipblasDestroy(cublasHandle));

  VERIFY_CUDA_CALL(hipMemcpy(reinterpret_cast<void *>(resultHost.data()),
                              reinterpret_cast<void *>(devC), totalBytes,
                              hipMemcpyDeviceToHost));

  VERIFY_CUDA_CALL(hipFree(reinterpret_cast<void *>(devA)));
  VERIFY_CUDA_CALL(hipFree(reinterpret_cast<void *>(devB)));
  VERIFY_CUDA_CALL(hipFree(reinterpret_cast<void *>(devC)));

  return resultHost;
}
