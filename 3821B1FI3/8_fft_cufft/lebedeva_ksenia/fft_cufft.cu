#include "hip/hip_runtime.h"
// Copyright (c) 2024 Lebedeva Ksenia
#include "fft_cufft.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>

#include <cstdlib>
#include <vector>

__global__ void myNormKernel(float* input, int size, float norm) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        input[i] *= norm;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    const int size = input.size();
    std::vector<float> output(size);

    int n = (size / batch) >> 1;
    int sizeInBytes = sizeof(hipfftComplex) * n * batch;

    hipfftComplex* data;
    hipMalloc(&data, sizeInBytes);
    hipMemcpy(data, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

    hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    float norm = 1.0f / static_cast<float>(n);
    myNormKernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(data), size, norm);  // hipfftComplex = [float, float]

    hipMemcpy(output.data(), data, sizeInBytes, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(data);

    return output;
}
