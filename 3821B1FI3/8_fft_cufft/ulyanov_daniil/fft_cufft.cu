#include "hip/hip_runtime.h"
// Copyright (c) 2024 Ulyanov Daniil

#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include ""
#include "fft_cufft.h"

#define VERIFY_CUDA_CALL(call)                                                 \
  {                                                                            \
    auto errorCode = call;                                                     \
    if (errorCode != hipSuccess) {                                            \
      std::cerr << "\033[1;31mCUDA Error:\033[0m ";                            \
      std::cerr << hipGetErrorString(errorCode) << '\n';                      \
      std::cerr << "Error Code: " << static_cast<int>(errorCode) << '\n';      \
      std::cerr << "Location: " << __FILE__ << " (" << __LINE__ << ")\n";      \
      std::exit(static_cast<int>(errorCode));                                  \
    }                                                                          \
  }

#define VERIFY_CUFFT_CALL(call)                                                \
  {                                                                            \
    auto cufftStatus = call;                                                   \
    if (cufftStatus != HIPFFT_SUCCESS) {                                        \
      std::cerr << "\033[1;31mcuFFT Error:\033[0m ";                           \
      std::cerr << static_cast<int>(cufftStatus) << '\n';                      \
      std::cerr << "Location: " << __FILE__ << " (" << __LINE__ << ")\n";      \
      std::exit(static_cast<int>(cufftStatus));                                \
    }                                                                          \
  }

__global__ void apply_normalization(float *data, size_t length, float factor) {
  size_t idx =
      static_cast<size_t>(blockDim.x) * static_cast<size_t>(blockIdx.x) +
      static_cast<size_t>(threadIdx.x);
  if (idx < length) {
    data[idx] *= factor;
  }
}

std::vector<float> FffCUFFT(const std::vector<float> &input, int batch) {
  hipDeviceProp_t deviceProperties{};
  VERIFY_CUDA_CALL(hipGetDeviceProperties(&deviceProperties, 0));

  if (input.empty())
    return {};

  auto totalSize = input.size();
  auto itemsPerBatch = totalSize / batch >> 1;
  auto byteCount = sizeof(hipfftComplex) * itemsPerBatch * batch;
  auto maxThreads = deviceProperties.maxThreadsPerBlock;
  auto totalBlocks = (totalSize + maxThreads - 1) / maxThreads;
  std::vector<float> result(totalSize);

  hipfftComplex *deviceSignal = nullptr;
  VERIFY_CUDA_CALL(hipMalloc(&deviceSignal, byteCount));
  VERIFY_CUDA_CALL(hipMemcpy(reinterpret_cast<void *>(deviceSignal),
                              reinterpret_cast<const void *>(input.data()),
                              byteCount, hipMemcpyHostToDevice));

  hipfftHandle fftHandle{};
  VERIFY_CUFFT_CALL(hipfftPlan1d(&fftHandle, itemsPerBatch, HIPFFT_C2C, batch));
  VERIFY_CUFFT_CALL(
      hipfftExecC2C(fftHandle, deviceSignal, deviceSignal, HIPFFT_FORWARD));
  VERIFY_CUFFT_CALL(
      hipfftExecC2C(fftHandle, deviceSignal, deviceSignal, HIPFFT_BACKWARD));
  VERIFY_CUFFT_CALL(hipfftDestroy(fftHandle));

  apply_normalization<<<totalBlocks, maxThreads>>>(
      reinterpret_cast<float *>(deviceSignal), totalSize, 1.0f / itemsPerBatch);
  VERIFY_CUDA_CALL(hipDeviceSynchronize());
  VERIFY_CUDA_CALL(hipGetLastError());

  VERIFY_CUDA_CALL(hipMemcpy(reinterpret_cast<void *>(result.data()),
                              reinterpret_cast<void *>(deviceSignal), byteCount,
                              hipMemcpyDeviceToHost));
  VERIFY_CUDA_CALL(hipFree(reinterpret_cast<void *>(deviceSignal)));

  return result;
}
