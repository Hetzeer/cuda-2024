#include "hip/hip_runtime.h"
// Copyright (c) 2024 Bakhtiarov Alexander

#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "naive_gemm_cuda.h"

#define BLOCK_SIZE 32

__global__ void NaiveGemmKernel(const float* a, const float* b, float* c,
                                 const size_t size)
{
    constexpr auto blockSize = BLOCK_SIZE;
    __shared__ float sharedA[blockSize][blockSize];
    __shared__ float sharedB[blockSize][blockSize];

    size_t iIdx = blockIdx.y * blockSize + threadIdx.y;
    size_t jIdx = blockIdx.x * blockSize + threadIdx.x;

    float result = 0.0f;

    for (size_t k = 0; k < size; k += blockSize) {

        if (jIdx < size && (threadIdx.y + k)  < size) {
            sharedB[threadIdx.y][threadIdx.x] = __ldg(&b[(threadIdx.y + k) * size + jIdx]);
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (iIdx < size && (threadIdx.x + k) < size) {
            sharedA[threadIdx.y][threadIdx.x] = __ldg(&a[iIdx * size + threadIdx.x + k]);
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (size_t l = 0; l < blockSize; ++l) {
            result += sharedA[threadIdx.y][l] * sharedB[l][threadIdx.x];
        }

        __syncthreads();
    }

    if (iIdx < size && jIdx < size) {
        c[iIdx * size + jIdx] = result;
    }
}


std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b, int size) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  auto countElem = size * size;
  std::vector<float> output(countElem);
  auto sizeInBytes = countElem * sizeof(float);

  dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
  auto t = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 numBlocks(t, t);

  float *aDev = nullptr;
  hipMalloc(&aDev, sizeInBytes);

  float *bDev = nullptr;
  hipMalloc(&bDev, sizeInBytes);

  float *cDev = nullptr;
  hipMalloc(&cDev, sizeInBytes);

  hipMemcpy(aDev, a.data(), sizeInBytes, hipMemcpyHostToDevice);
  hipMemcpy(bDev, b.data(), sizeInBytes, hipMemcpyHostToDevice);

  NaiveGemmKernel<<<numBlocks, threadsPerBlock>>>(aDev, bDev, cDev, size);

  hipDeviceSynchronize();
  hipMemcpy(output.data(), cDev, sizeInBytes, hipMemcpyDeviceToHost);

  hipFree(cDev);
  hipFree(bDev);
  hipFree(aDev);

  return output;
}
