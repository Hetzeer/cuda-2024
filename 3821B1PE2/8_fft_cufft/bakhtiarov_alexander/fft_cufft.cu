//Copyright 2024 Bakhtiarov Alexander

#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <stdexcept>
#include <iostream>
#include <hip/hip_runtime.h>

// Helper function to check CUDA errors
void checkCudaError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA error occurred.");
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    // Check if the input size is valid
    if (input.size() % 2 != 0) {
        throw std::invalid_argument("Input size must be even (real and imaginary pairs).");
    }

    int n = input.size() / (2 * batch);  // n is the number of complex elements per signal

    // Step 1: Allocate memory for the input and output on the device
    hipfftComplex* d_input;
    hipfftComplex* d_output;

    checkCudaError(hipMalloc(&d_input, sizeof(hipfftComplex) * n * batch));
    checkCudaError(hipMalloc(&d_output, sizeof(hipfftComplex) * n * batch));

    // Step 2: Copy input data from host to device
    std::vector<hipfftComplex> h_input(n * batch);
    for (int i = 0; i < n * batch; ++i) {
        h_input[i].x = input[2 * i];      // real part
        h_input[i].y = input[2 * i + 1];  // imaginary part
    }

    checkCudaError(hipMemcpy(d_input, h_input.data(), sizeof(hipfftComplex) * n * batch, hipMemcpyHostToDevice));

    // Step 3: Create cuFFT plan for 1D complex-to-complex transform
    hipfftHandle plan;
    if (hipfftPlan1d(&plan, n, HIPFFT_C2C, batch) != HIPFFT_SUCCESS) {
        throw std::runtime_error("CUFFT plan creation failed.");
    }

    // Step 4: Perform the forward FFT (C2C transform)
    if (hipfftExecC2C(plan, d_input, d_output, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        throw std::runtime_error("CUFFT forward transform failed.");
    }

    // Step 5: Perform the inverse FFT (C2C transform)
    if (hipfftExecC2C(plan, d_output, d_output, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        throw std::runtime_error("CUFFT inverse transform failed.");
    }

    // Step 6: Normalize the result by dividing by n
    hipMemcpy(h_input.data(), d_output, sizeof(hipfftComplex) * n * batch, hipMemcpyDeviceToHost);

    for (int i = 0; i < n * batch; ++i) {
        h_input[i].x /= n;
        h_input[i].y /= n;
    }

    // Step 7: Copy the result back to a flat vector of floats (real, imaginary pairs)
    std::vector<float> result(2 * n * batch);
    for (int i = 0; i < n * batch; ++i) {
        result[2 * i] = h_input[i].x;
        result[2 * i + 1] = h_input[i].y;
    }

    // Step 8: Clean up
    hipFree(d_input);
    hipFree(d_output);
    hipfftDestroy(plan);

    return result;
}