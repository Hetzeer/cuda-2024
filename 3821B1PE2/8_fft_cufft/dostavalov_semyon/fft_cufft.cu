#include "hip/hip_runtime.h"
// Copyright (c) 2024 Dostavalov Semyon

#include <iostream>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_cufft.h"


__global__ void normalizeKernel(float* __restrict__ input, int size, float normalizationFactor) {
    const int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
    const int vectorIndex = threadIndex / 4;

    if (vectorIndex < size / 4) {
        float4* inputVector = reinterpret_cast<float4*>(input);
        float4 data = __ldg(&inputVector[vectorIndex]);
        data.x *= normalizationFactor;
        data.y *= normalizationFactor;
        data.z *= normalizationFactor;
        data.w *= normalizationFactor;
        inputVector[vectorIndex] = data;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) 
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int size = input.size();
    std::vector<float> output(size);

    const int elemPerBatch = size / batch >> 1;
    const int sizeInBytes = sizeof(hipfftComplex) * elemPerBatch * batch;
    const int threadsPerBlock = deviceProp.maxThreadsPerBlock;
    const int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    hipfftComplex* signal;
    hipMalloc(&signal, sizeInBytes);
    hipMemcpy(signal, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipfftHandle handle;
    hipfftPlan1d(&handle, elemPerBatch, HIPFFT_C2C, batch);
    hipfftExecC2C(handle, signal, signal, HIPFFT_FORWARD);
    hipfftExecC2C(handle, signal, signal, HIPFFT_BACKWARD);

    normalizeKernel<<<numBlocks, threadsPerBlock>>>(reinterpret_cast<float*>(signal), size, 1.0f / elemPerBatch);

    hipMemcpy(output.data(), signal, sizeInBytes, hipMemcpyDeviceToHost);

    hipfftDestroy(handle);
    hipFree(signal);

    return output;
}
